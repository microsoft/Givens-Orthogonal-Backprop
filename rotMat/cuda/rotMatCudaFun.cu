#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <utility>
#include <functional>
#include <vector>

#define ThreadsPerBlockForward 128
#define ThreadsPerBlockBackward 256

using namespace torch::indexing;

__device__ __forceinline__ std::pair<const int, const int> determineRowIndexPair(
  const int blockIndex,
  const size_t Ntilde,
  const int tournamentStep)
{
  const int n = Ntilde - 1;

  // Determine i,j from block index
  int i = blockIndex==0 ? 0 : blockIndex  + tournamentStep;
  if (i >= Ntilde)
  {
    i -= n;
  }

  int j = n - blockIndex + tournamentStep;
  if (j >= Ntilde)
  {
    j -= n;
  }

  return i > j ? std::make_pair(j,i) : std::make_pair(i,j);
}

__device__ __forceinline__ bool areRowIndicesOutOfRange(const int i, const int j, const int deadIndex, const int dMax)
{
  // check if the coordinates are out of range or equal dummy coordinate (dummy exists when N is odd)
  return j == deadIndex || (i > dMax && j > dMax);
}

 __global__ void updateGivensElements(
  const float* const __restrict__ C,
  const float* const __restrict__ S,
  at::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> U,
  const int deadIndex,
  const size_t Ntilde,
  const int dMax,
  const int tournamentStep)
{
  // If transpose k works on rows; otherwise on columns
  const int k = threadIdx.y + blockDim.y*blockIdx.y;
  const int N = U.size(1);
  if (k >= N)
  {
    return;
  }

  auto rowIndices = determineRowIndexPair(blockIdx.x, Ntilde, tournamentStep);
  const int i = rowIndices.first;
  const int j = rowIndices.second;

  if (areRowIndicesOutOfRange(i, j, deadIndex, dMax))
  {
    return;
  }

  const int thetaIndex = i*N - (i+2)*(i+1)/2 + j;
  const float cij = C[thetaIndex];
  const float sij = S[thetaIndex];

  // Apply Givens: Update U's offsets
  const float Ui = U[i][k];
  const float Uj = U[j][k];

  U[i][k] = Ui*cij - Uj*sij;
  U[j][k] = Ui*sij + Uj*cij;
}

__device__ void warpReduceAtBackward(volatile float* sdata, int tid)
{
  if (ThreadsPerBlockBackward >= 64)  sdata[tid] += sdata[tid + 32];
  if (ThreadsPerBlockBackward >= 32) sdata[tid] += sdata[tid + 16];
  if (ThreadsPerBlockBackward >= 16) sdata[tid] += sdata[tid + 8];
  if (ThreadsPerBlockBackward >= 8) sdata[tid] += sdata[tid + 4];
  if (ThreadsPerBlockBackward >= 4) sdata[tid] += sdata[tid + 2];
  if (ThreadsPerBlockBackward >= 2) sdata[tid] += sdata[tid + 1];
}

__global__ void setJVP(
  at::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> M,
  at::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits>UfTrans,
  const float* const __restrict__ C,
  const float* const __restrict__ S,
  float* __restrict__ JVP,
  const int deadIndex,
  const int Ntilde,
  const int dMax,
  const int tournamentStep)
{

  extern __shared__ float sA[];

  // k is the column index of M and the row index of Uf, to set col of A
  const int tid = threadIdx.y;
  const int k = tid + blockDim.y*blockIdx.y;
  const int N = UfTrans.size(1);
  if (k >= N)
  {
    sA[tid] = 0;
    return;
  }

  auto rowIndices = determineRowIndexPair(blockIdx.x, Ntilde, tournamentStep);
  int i = rowIndices.first;
  int j = rowIndices.second;

  if (areRowIndicesOutOfRange(i, j, deadIndex, dMax))
  {
    sA[tid] = 0;
    return;
  }

   __syncthreads();
  const int thetaIndex = i*N - (i+2)*(i+1)/2 + j;
  const float cij = C[thetaIndex];
  const float sij = S[thetaIndex];

  // Apply Givens: Update U's offsets
  const float Ui = UfTrans[i][k];
  const float Uj = UfTrans[j][k];

  const float newUfTik = Ui*cij - Uj*sij;
  UfTrans[i][k] = newUfTik;

  const float newUfTjk = Ui*sij + Uj*cij;
  UfTrans[j][k] = newUfTjk;

  // Repeat for M
  const float Mi = M[i][k];
  const float Mj = M[j][k];

  const float newMik = Mi*cij - Mj*sij;
  M[i][k] = newMik;

  const float newMjk = Mi*sij + Mj*cij;
  M[j][k] = newMjk;

  // Set A, skip a write if possible can
  sA[tid] = newMik * newUfTjk - newMjk * newUfTik;
  __syncthreads();

  // Reduce
  if (ThreadsPerBlockBackward == 1024) {
    if (tid < 512) { sA[tid] += sA[tid + 512]; } __syncthreads();}
  if (ThreadsPerBlockBackward >= 512) {
    if (tid < 256) { sA[tid] += sA[tid + 256]; } __syncthreads();}
  if (ThreadsPerBlockBackward >= 256) {
    if (tid < 128) { sA[tid] += sA[tid + 128]; } __syncthreads(); }
  if (ThreadsPerBlockBackward >= 128) {
    if (tid < 64) { sA[tid] += sA[tid + 64]; } __syncthreads(); }
  if(tid <32) warpReduceAtBackward(sA,tid);
  
  if (tid == 0) atomicAdd(&JVP[thetaIndex], sA[0]);
}

std::tuple<int64_t, int64_t, int64_t> determineRotMatConstants(const size_t nThetas, const size_t N)
{
  auto maxPairs = N*(N-1)/2;

  // if nThetas == maxPairs
  auto dMax = N-1;
  if (nThetas < maxPairs)
  {
    auto K = int(1 + sqrt(1 - 4*(2*nThetas - N*(N-1)))) / 2;
    dMax -= K;
  }

  // Handle odd N; in that case Ntilde is the even augmented dimension
  int64_t deadIndex = -1;
  auto Ntilde = N;
  if (N % 2 != 0)
  {
    Ntilde += 1;
    deadIndex = Ntilde-1;
  }

  return std::make_tuple(dMax, deadIndex, Ntilde);
}

torch::Tensor rotMatForwardCuda(torch::Tensor thetas, int64_t N)
{
  auto rotMatConstants = determineRotMatConstants(thetas.size(0), N);
  auto dMax = std::get<0>(rotMatConstants);
  auto deadIndex = std::get<1>(rotMatConstants);
  auto Ntilde = std::get<2>(rotMatConstants);

  // Set U same device and type as thetas
  auto tensOptions = torch::TensorOptions()
    .dtype(thetas.dtype())
    .device(thetas.device());

  auto U = torch::eye(N, tensOptions);
  auto C = torch::cos(thetas.detach());
  auto S = torch::sin(thetas.detach());

  // CUDA grid: blocks of size: Ntilde/2 x ceil(N/nThreads)
  const int nBlocksY = N/ThreadsPerBlockForward + (N % ThreadsPerBlockForward != 0);
  const int nBlocksX = Ntilde / 2;

  const dim3 blocks(nBlocksX, nBlocksY);
  const dim3 threads(1, ThreadsPerBlockForward);

  // The circle-method is used to generate round-robin sequences per block (equivalent to scheduling round-robin sports tournaments)
  // 'tournamentStep' refers to to the current turn of the tournament, where all updates are executed in parallel. There are n-1 steps
  for (int64_t tournamentStep=Ntilde-2; tournamentStep>=0; tournamentStep--)
  {
      updateGivensElements<<<blocks,threads>>>(
        C.data_ptr<float>(),
        S.data_ptr<float>(),
        U.packed_accessor32<float, 2, at::RestrictPtrTraits>(),
        deadIndex, Ntilde, dMax, tournamentStep);
  }

  return U;
}

torch::Tensor rotMatBackwardCuda(
  torch::Tensor thetas,
  torch::Tensor U,
  torch::Tensor G)
{
  auto N = U.size(0);
  auto rotMatConstants = determineRotMatConstants(thetas.size(0), N);
  auto dMax = std::get<0>(rotMatConstants);
  auto deadIndex = std::get<1>(rotMatConstants);
  auto Ntilde = std::get<2>(rotMatConstants);

  // In rotMatForwardCuda, U is given these same properties
  auto tensOptions = torch::TensorOptions()
    .dtype(thetas.dtype())
    .device(thetas.device());

  auto M = G.t().contiguous().detach();
  auto UfTrans = U.t().contiguous().detach();

  auto C = torch::cos(thetas.detach());
  auto S = torch::sin(thetas.detach());
  auto JVP = torch::zeros_like(thetas, tensOptions);

  // CUDA grid: blocks of size: Ntilde/2 x ceil(N/nThreads)
  const int nBlocksX = Ntilde / 2;
  const int nBlocksY = N/ThreadsPerBlockBackward + (N % ThreadsPerBlockBackward != 0);

  const dim3 blocks(nBlocksX, nBlocksY);
  const dim3 threads(1, ThreadsPerBlockBackward);

  // The circle-method is used to generate round-robin sequences per block (equivalent to scheduling round-robin sports tournaments)
  // 'tournamentStep' refers to to the current turn of the tournament, where all updates are executed in parallel. here are n-1 steps
  for (int tournamentStep=Ntilde-2; tournamentStep>=0; tournamentStep--)
  {
    // Set A els
    setJVP<<<blocks,threads, sizeof(float) * ThreadsPerBlockBackward>>>(
        M.packed_accessor32<float, 2, at::RestrictPtrTraits>(),
        UfTrans.packed_accessor32<float, 2, at::RestrictPtrTraits>(),
        C.data_ptr<float>(),
        S.data_ptr<float>(),
        JVP.data_ptr<float>(),
        deadIndex, Ntilde, dMax, tournamentStep);
  }
  
  return JVP;
}
