#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <utility>
#include <functional>
#include <vector>

//using namespace torch::indexing;

#define MaximumThreadsPerBlock 1024
#define WarpSize 32

#define ThreadsPerRowForward 64
#define ThreadsPerRowBackward 128

// DELETE FORWARD SUFFIX, SAME FOR FORWARD AND BACKWARD
#define InterTeamRRThreadsPerBlockForward MaximumThreadsPerBlock
#define InterTeamBlockDepthForward (int)(InterTeamRRThreadsPerBlockForward/256)
#define InterTeamBlockWidthForward (int)(InterTeamRRThreadsPerBlockForward/InterTeamBlockDepthForward)

// Current implementation dictates InterTeamBlockDepthForward to be 2 *IntraTeamBlockDepthForward
#define IntraTeamRRThreadsPerBlockForward (int)(MaximumThreadsPerBlock)
#define IntraTeamBlockDepthForward (int)(InterTeamBlockDepthForward/2)
#define IntraTeamBlockWidthForward (int)(IntraTeamRRThreadsPerBlockForward/IntraTeamBlockDepthForward)


// https://stackoverflow.com/questions/12626096/why-has-atomicadd-not-been-implemented-for-doubles
// https://stackoverflow.com/questions/37566987/cuda-atomicadd-for-doubles-definition-error
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(
  double* address, 
  double val)
{
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif

/************************* HELPER FUNCTIONS *********************************/

int getTeamSizeForTesting()
{
  return InterTeamBlockDepthForward;
}

__device__ __forceinline__ std::pair<const int, const int> determineRowIndexPair(
  const int blockIndex,
  const int Ntilde,
  const int tournamentStep)
{
  const int n = Ntilde - 1;

  // Determine i,j from block index
  int i = blockIndex==0 ? 0 : blockIndex  + tournamentStep;
  if (i >= Ntilde)
  {
    i -= n;
  }

  int j = n - blockIndex + tournamentStep;
  if (j >= Ntilde)
  {
    j -= n;
  }

  return i > j ? std::make_pair(j,i) : std::make_pair(i,j);
}

__device__ __forceinline__ bool areRowIndicesOutOfRange(
  const int i, 
  const int j, 
  const int firstDummyIndex, 
  const int dMax)
{
  // check if the coordinates are out of range or equal dummy coordinate (dummy exists when N is odd)
  return j >= firstDummyIndex || (i > dMax && j > dMax);
}

void incrementIfNotEven(int &Ntilde)
{
  if (Ntilde % 2 != 0)
  {
    Ntilde += 1;
  }
}

std::tuple<int, int, int> determineRotMatConstants(const int nThetas, const int N)
{
  auto dMax = N-1; // If nThetas == maxPairs
  if (nThetas < N*(N-1)/2)
  {
    dMax -= 1 + int(sqrt(1 - 4*(2*nThetas - N*(N-1)))) / 2;
  }

  // Handle odd N; in that case Ntilde is the even augmented dimension
  int Ntilde = N;
  const int dummyIndex = Ntilde;
  incrementIfNotEven(Ntilde);

  return std::make_tuple(dMax, dummyIndex, Ntilde);
}

const dim3 prepareBlocksForTournament(const int B, const int &Ntilde, const int &width, const int &depth )
{
  const int nBlocksX = (B/width) + (B % width != 0);
  
  const int rotationsPerRound = Ntilde/2;
  const int nBlocksY = (rotationsPerRound / depth) + (rotationsPerRound % depth != 0); 
  
  const dim3 blocks(nBlocksX, nBlocksY);
  return blocks;
}



template <typename scalar_t>
  __device__ void blockReduceAtBackward(
    volatile scalar_t* sdata, 
    int tid)
{
  if (ThreadsPerRowBackward >= 64)  sdata[tid] += sdata[tid + 32];
  if (ThreadsPerRowBackward >= 32) sdata[tid] += sdata[tid + 16];
  if (ThreadsPerRowBackward >= 16) sdata[tid] += sdata[tid + 8];
  if (ThreadsPerRowBackward >= 8) sdata[tid] += sdata[tid + 4];
  if (ThreadsPerRowBackward >= 4) sdata[tid] += sdata[tid + 2];
  if (ThreadsPerRowBackward >= 2) sdata[tid] += sdata[tid + 1];
}


template <typename scalar_t>
  __device__ void blockReduceAtBackwardIntraTeamRR(
    volatile scalar_t* sdata,
    int tid)
{
  if (IntraTeamRRThreadsPerBlockForward >= 64)  sdata[tid] += sdata[tid + 32];
  if (IntraTeamRRThreadsPerBlockForward >= 32) sdata[tid] += sdata[tid + 16];
  if (IntraTeamRRThreadsPerBlockForward >= 16) sdata[tid] += sdata[tid + 8];
  if (IntraTeamRRThreadsPerBlockForward >= 8) sdata[tid] += sdata[tid + 4];
  if (IntraTeamRRThreadsPerBlockForward >= 4) sdata[tid] += sdata[tid + 2];
  if (IntraTeamRRThreadsPerBlockForward >= 2) sdata[tid] += sdata[tid + 1];
}

template <typename scalar_t>
  __device__ void blockReduceAtBackwardInterTeamRR(
    volatile scalar_t* sdata, 
    int tid)
{
  if (InterTeamRRThreadsPerBlockForward >= 64)  sdata[tid] += sdata[tid + 32];
  if (InterTeamRRThreadsPerBlockForward >= 32) sdata[tid] += sdata[tid + 16];
  if (InterTeamRRThreadsPerBlockForward >= 16) sdata[tid] += sdata[tid + 8];
  if (InterTeamRRThreadsPerBlockForward >= 8) sdata[tid] += sdata[tid + 4];
  if (InterTeamRRThreadsPerBlockForward >= 4) sdata[tid] += sdata[tid + 2];
  if (InterTeamRRThreadsPerBlockForward >= 2) sdata[tid] += sdata[tid + 1];
}

/************************* FORWARD PROPAGATION*******************************/
/****** USING THE CIRCLE METHOD FOR GENERATING THE ROUND ROBIN SEQUENCE *****/

 template <typename scalar_t>  
  __global__ void ApplyRoundRobinGivensRotationMatrix(
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> C,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> S,
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> U,
    const int dummyIndex,
    const int Ntilde,
    const int dMax,
    const int tournamentStep)
{
  const int k = threadIdx.y + blockDim.y*blockIdx.y;
  if (k >= U.size(1))
  {
    return;
  }

  auto rowIndices = determineRowIndexPair(blockIdx.x, Ntilde, tournamentStep);
  const int i = rowIndices.first;
  const int j = rowIndices.second;
  if (areRowIndicesOutOfRange(i, j, dummyIndex, dMax))
  {
    return;
  }

  const int N = U.size(0);
  const int thetaIndex = i*N - (i+2)*(i+1)/2 + j;
  const scalar_t cij = C[thetaIndex];
  const scalar_t sij = S[thetaIndex];

  // Apply Givens
  const scalar_t Ui = U[i][k];
  const scalar_t Uj = U[j][k];

  U[i][k] = Ui*cij - Uj*sij;
  U[j][k] = Ui*sij + Uj*cij;
}

torch::Tensor rotMatForwardCuda(torch::Tensor X, torch::Tensor thetas)
{
  const int N = X.size(0);
  auto rotMatConstants = determineRotMatConstants(thetas.size(0), N);
  auto dMax = std::get<0>(rotMatConstants);
  auto dummyIndex = std::get<1>(rotMatConstants);
  auto Ntilde = std::get<2>(rotMatConstants);

  auto C = torch::cos(thetas.detach());
  auto S = torch::sin(thetas.detach());

  const int nBlocksY = X.size(1)/ThreadsPerRowForward + (X.size(1)% ThreadsPerRowForward != 0);
  const dim3 blocks(Ntilde / 2, nBlocksY);
  const dim3 threads(1, ThreadsPerRowForward);

  // The circle-method is used to generate round-robin sequences per block (equivalent to scheduling round-robin sports tournaments)
  // 'tournamentStep' refers to to the current turn of the tournament, where all updates are executed in parallel. There are n-1 steps
  for (int tournamentStep=Ntilde-2; tournamentStep>=0; tournamentStep--)
  {
    AT_DISPATCH_FLOATING_TYPES(
      thetas.type(),
      "rotMatForwardCuda",
      ([&]{
        ApplyRoundRobinGivensRotationMatrix<scalar_t><<<blocks,threads>>>(
          C.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
          S.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
          X.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
          dummyIndex,Ntilde, dMax, tournamentStep);
      }));
  }

  return X;
}

/************************* BACKWARD PROPAGATION******************************/
// USING THE CIRCLE ROUND ROBIN TOURNAMENT FOR SEQUENCING GIVENS ROTATIONS

template <typename scalar_t> 
  __global__ void CalculateRoundRobinGivensThetaJVPs(
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> UX,
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> G,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> C,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> S,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> JVP,
    const int dummyIndex,
    const int Ntilde,
    const int dMax,
    const int tournamentStep)
{
  __shared__ scalar_t sA[ThreadsPerRowBackward];

  // k is the column index of M and the row index of Uf, to set col of A
  const int tid = threadIdx.y;
  const int k = tid + blockDim.y*blockIdx.y;
  if (k >= UX.size(1))
  {
    sA[tid] = 0;
    return;
  }

  auto rowIndices = determineRowIndexPair(blockIdx.x, Ntilde, tournamentStep);
  int i = rowIndices.first;
  int j = rowIndices.second;
  if (areRowIndicesOutOfRange(i, j, dummyIndex, dMax))
  {
    sA[tid] = 0;
    return;
  }

   __syncthreads();
  const int N = UX.size(0);
  const int thetaIndex = i*N - (i+2)*(i+1)/2 + j;
  const scalar_t cij = C[thetaIndex];
  const scalar_t sij = -1 * S[thetaIndex]; // Transpose of a Givens rotation has the signs of sij flipped

  // Apply Givens Transpose to G
  const scalar_t Gi = G[i][k];
  const scalar_t Gj = G[j][k];

  const scalar_t newGi = Gi*cij - Gj*sij;
  G[i][k] = newGi;

  const scalar_t newGj = Gi*sij + Gj*cij;
  G[j][k] = newGj;

  // Repeat for UX
  const scalar_t UXi = UX[i][k];
  const scalar_t UXj = UX[j][k];

  const scalar_t newUXi = UXi*cij - UXj*sij;
  UX[i][k] = newUXi;

  const scalar_t newUXj = UXi*sij + UXj*cij;
  UX[j][k] = newUXj;

  sA[tid] = newUXi * newGj - newUXj * newGi;
  __syncthreads();

  // Reduce
  if (ThreadsPerRowBackward == 1024) {
    if (tid < 512) { sA[tid] += sA[tid + 512]; } __syncthreads();}
  if (ThreadsPerRowBackward >= 512) {
    if (tid < 256) { sA[tid] += sA[tid + 256]; } __syncthreads();}
  if (ThreadsPerRowBackward >= 256) {
    if (tid < 128) { sA[tid] += sA[tid + 128]; } __syncthreads(); }
  if (ThreadsPerRowBackward >= 128) {
    if (tid < 64) { sA[tid] += sA[tid + 64]; } __syncthreads(); }
  if(tid < 32) blockReduceAtBackward(sA, tid);
  
  if (tid == 0)  atomicAdd(&JVP[thetaIndex], sA[tid]);
}

std::pair<torch::Tensor, torch::Tensor> rotMatBackwardCuda(
  torch::Tensor thetas,
  torch::Tensor UX,
  torch::Tensor G)
{

  auto rotMatConstants = determineRotMatConstants(thetas.size(0), UX.size(0));
  auto dMax = std::get<0>(rotMatConstants);
  auto dummyIndex = std::get<1>(rotMatConstants);
  auto Ntilde = std::get<2>(rotMatConstants);

  auto C = torch::cos(thetas.detach());
  auto S = torch::sin(thetas.detach());

  auto thetasTensorOptions = torch::TensorOptions().dtype(thetas.dtype()).device(thetas.device());
  auto JVP = torch::zeros_like(thetas, thetasTensorOptions);

  auto B = UX.size(1);
  const int nBlocksY = B/ThreadsPerRowBackward + (B % ThreadsPerRowBackward != 0);
  const dim3 blocks(Ntilde / 2, nBlocksY);
  const dim3 threads(1, ThreadsPerRowBackward);

  // The circle-method is used to generate round-robin sequences per block (equivalent to scheduling round-robin sports tournaments)
  // 'tournamentStep' refers to to the current turn of the tournament, where all updates are executed in parallel. here are n-1 steps
  for (int tournamentStep=0; tournamentStep<=Ntilde-2; tournamentStep++)
  {
    AT_DISPATCH_FLOATING_TYPES(
      thetas.type(),
      "rotMatBackwardCuda",
      ([&]{
        CalculateRoundRobinGivensThetaJVPs<scalar_t><<<blocks,threads>>>(
          UX.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
          G.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
          C.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
          S.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
          JVP.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
          dummyIndex, Ntilde, dMax, tournamentStep);
      }));
  }
  
  return std::make_pair(G, JVP);
}

/************************* FORWARD PROPAGATION*******************************/
// USING THE TEAM ROUND ROBIN TOURNAMENT FOR SEQUENCING GIVENS ROTATIONS

template <typename scalar_t>
  __global__ void PlayIntraTeamTournament(
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> C,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> S,
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> X,
    const int Ntilde,
    const int dummyIndex,
    const int dMax)
{
  // If transpose k works on rows; otherwise on columns
  const int tid = threadIdx.x;
  const int tidY = threadIdx.y;
  const int k = tid + blockDim.x*blockIdx.x; 

  if (k >= X.size(1))
  {
    return;
  }

  int playerCountInBlock = IntraTeamBlockDepthForward *2;
  //if( k == 0 )  printf("init player count int block FORWARD: %d\n",  playerCountInBlock);

  int blockStart = playerCountInBlock * blockIdx.y;

  //if( k == 0 )  printf("block start %d NTilde %d FORWARD: \n",  blockStart, Ntilde);

  if (playerCountInBlock > Ntilde-blockStart)
  {
    playerCountInBlock = Ntilde-blockStart;
  }

  if (tidY*2 >= playerCountInBlock)
  {
    return;
  }

  const int N = X.size(0);
  int i, j, thetaIndex;
  scalar_t cij, sij, Xi, Xj;
  
  //if( k == 0 )  printf("dummy id %d dMax %d  player count int block: %d\n", dummyIndex, dMax, playerCountInBlock);
  for (int tournamentStep=0; tournamentStep<=playerCountInBlock-2; tournamentStep++)
  {
    auto rowIndices = determineRowIndexPair(tidY, playerCountInBlock, tournamentStep);
    i = blockStart + rowIndices.first;
    j = blockStart + rowIndices.second;
    
    //if( k == 0 ) printf("blockId %d thread id %d  tournamentStep: %d  i,j => %d %d INTRA FORWARD got one! ->  S %.6f C %.6f \n ",blockIdx.y, tidY, tournamentStep, i, j, sij, cij);

    if (areRowIndicesOutOfRange(i, j, dummyIndex, dMax))
    {
      __syncthreads();
      continue;
    }

    thetaIndex = i*N - (i+2)*(i+1)/2 + j;
    cij = C[thetaIndex];
    sij = S[thetaIndex];
    
    Xi = X[i][k];
    Xj = X[j][k];
    //if( tid == 0 ) printf("%d %d got one! -> S %.6f C %.6f \n", i, j, sij, cij);

    X[i][k] = Xi*cij - Xj*sij;
    X[j][k] = Xi*sij + Xj*cij;
    __syncthreads();
  }
}

template <typename scalar_t> __global__ void PlayTeamTournamentMatch(
  at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> C,
  at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> S,
  at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> X,
  const int dummyIndex,
  const int dMax,
  const int teamCount,
  const int dummyTeamIndex,
  const int tournamentStep)
{
  // If transpose k works on rows; otherwise on columns
  const int col = threadIdx.x + blockDim.x*blockIdx.x;
  if (col >= X.size(1))
  {
    return;
  }

  auto matchedTeams = determineRowIndexPair(blockIdx.y, teamCount, tournamentStep);
  if (matchedTeams.second == dummyTeamIndex)  return;

  const int playerCountPerTeam = blockDim.y;
  const int i =  playerCountPerTeam * matchedTeams.first + threadIdx.y; // home team
  scalar_t Xi = X[i][col];
  
  const int jStart = playerCountPerTeam * matchedTeams.second; //visiting team
  const int jEnd = jStart + playerCountPerTeam;
  int j = jStart + threadIdx.y;
  
  const int N = X.size(0);
  for (int step =0; step < playerCountPerTeam; step++, j++)
  {
    if (j >= jEnd)
    {
      j -= playerCountPerTeam;
    }
    
    if (areRowIndicesOutOfRange(i, j, dummyIndex, dMax))
    {
      __syncthreads();
      continue;
    }

    const int thetaIndex = i*N - (i+2)*(i+1)/2 + j;
    const scalar_t cij = C[thetaIndex];
    const scalar_t sij = S[thetaIndex];

    // Apply Givens: Update U's offsets
    const scalar_t Xj = X[j][col];

    //if( col == 0 ) printf("%d %d got one! ->  S %.6f C %.6f \n ", i, j, sij, cij);

    // must update uj before updating ui
    X[j][col] = Xi*sij + Xj*cij;
    Xi = Xi*cij - Xj*sij;

    __syncthreads();
  }

  X[i][col] = Xi;
}


bool ScheduleIntraTeamTournaments(
  torch::Tensor C, 
  torch::Tensor S, 
  torch::Tensor X,
  std::tuple<int, int, int> rotMatConstants)
{
  auto dMax = std::get<0>(rotMatConstants);
  auto dummyIndex = std::get<1>(rotMatConstants);
  auto Ntilde = std::get<2>(rotMatConstants);

  const dim3 blocks = prepareBlocksForTournament(X.size(1), Ntilde, IntraTeamBlockWidthForward, IntraTeamBlockDepthForward);
  const dim3 threads(IntraTeamBlockWidthForward, IntraTeamBlockDepthForward);

  AT_DISPATCH_FLOATING_TYPES(
    C.type(),
    "rotMatForwardCuda",
    ([&]{ PlayIntraTeamTournament<scalar_t><<<blocks,threads>>>(
      C.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
      S.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
      X.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
      Ntilde, dummyIndex, dMax);}));

  return blocks.y == 1;
}

void ScheduleInterTeamTournament(
  torch::Tensor C, 
  torch::Tensor S, 
  torch::Tensor X,
  std::tuple<int, int, int> rotMatConstants)
{
  auto dMax = std::get<0>(rotMatConstants);
  auto dummyIndex = std::get<1>(rotMatConstants);
  auto Ntilde = std::get<2>(rotMatConstants);
  
  const dim3 blocks = prepareBlocksForTournament(X.size(1), Ntilde, InterTeamBlockWidthForward, InterTeamBlockDepthForward);
  const dim3 threads(InterTeamBlockWidthForward, InterTeamBlockDepthForward);

  int teamCount = (Ntilde/InterTeamBlockDepthForward) + (Ntilde%InterTeamBlockDepthForward != 0);
  const int dummyTeamIndex = teamCount;
  incrementIfNotEven(teamCount);
  
  for (int tournamentStep=teamCount-2; tournamentStep>=0; tournamentStep--)
  {
    AT_DISPATCH_FLOATING_TYPES(
      C.type(),
      "rotMatForwardCuda",
      ([&]{ PlayTeamTournamentMatch<scalar_t><<<blocks,threads>>>(
        C.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
        S.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
        X.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
        dummyIndex, dMax, teamCount, dummyTeamIndex, tournamentStep);}));
  }
}


torch::Tensor rotMatForwardCudaTeamRR(torch::Tensor X, torch::Tensor thetas)
{
  auto constants = determineRotMatConstants(thetas.size(0), X.size(0));
  auto C = torch::cos(thetas.detach());
  auto S = torch::sin(thetas.detach());
  
  //bool allThetasFitToOneTeam = ScheduleIntraTeamTournaments(C, S, X, constants);
  //if (allThetasFitToOneTeam) return X;
  ScheduleInterTeamTournament(C, S, X, constants);
  
  return X;
}

/************************* BACKWARD PROPAGATION*******************************/
// USING THE TEAM ROUND ROBIN TOURNAMENT FOR SEQUENCING GIVENS ROTATIONS

template <typename scalar_t> 
  __global__ void PlayTeamTournamentMatchForThetaGrad(
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> UX,
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> G,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> C,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> S,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> JVP,
    const int dummyIndex,
    const int dMax,
    const int teamCount,
    const int dummyTeamIndex,
    const int tournamentStep)
{
  __shared__ scalar_t sAGridForBlock[InterTeamBlockDepthForward][InterTeamBlockWidthForward];
  scalar_t* sA = sAGridForBlock[threadIdx.y];

  // If transpose k works on rows; otherwise on columns
  const int k = threadIdx.x + blockDim.x*blockIdx.x;
  const int tid = threadIdx.x;
  if (k >= UX.size(1)) // do we need a tidyY check here?
  {
    sA[tid] = 0;
    return;
  }

  auto matchedTeams = determineRowIndexPair(blockIdx.y, teamCount, tournamentStep);
  if (matchedTeams.second == dummyTeamIndex)  return;

  const int playerCountPerTeam = blockDim.y;
  const int i =  playerCountPerTeam * matchedTeams.first + threadIdx.y; // home team
  scalar_t UXi = UX[i][k];
  scalar_t Gi = G[i][k];

  const int jStart = playerCountPerTeam * matchedTeams.second; //visiting team
  const int jEnd = jStart + playerCountPerTeam;
  int j = jStart + threadIdx.y - 1;
  
  const int N = UX.size(0);
  int thetaIndex;
  scalar_t cij, sij, UXj, newUXi, newUXj, Gj, newGi, newGj;
  for (int step =0; step < playerCountPerTeam; step++, j--)
  {
    if (j < jStart)
    {
      j += playerCountPerTeam;
    }
    
    if (areRowIndicesOutOfRange(i, j, dummyIndex, dMax))
    {
      sA[tid] = 0;
      __syncthreads();
      continue;
    }

    thetaIndex = i*N - (i+2)*(i+1)/2 + j;
    cij = C[thetaIndex];
    sij = -S[thetaIndex];

    //if( k == 0 ) printf("%d %d NACKWARD got one! ->  S %.6f C %.6f \n ", i, j, sij, cij);

    // Apply Givens: Update U's offsets
    UXj = UX[j][k];
    newUXj = UXi*sij + UXj*cij; // must update j before updating i
    UXi = UXi*cij - UXj*sij; 

    Gj = G[j][k];
    newGj = Gi*sij + Gj*cij; // must update j before updating i
    Gi = Gi*cij - Gj*sij;

    UX[j][k] = newUXj; 
    G[j][k] = newGj;

    auto res = newUXi * newGj - newUXj * newGi;
    sA[tid] = res;
    atomicAdd(&JVP[thetaIndex], res);
    __syncthreads();

    /* Reduce
    if (InterTeamBlockWidthForward == 1024) {
      if (tid < 512) { sA[tid] += sA[tid + 512]; } __syncthreads(); }
    if (InterTeamBlockWidthForward >= 512) {
      if (tid < 256) { sA[tid] += sA[tid + 256]; } __syncthreads(); }
    if (InterTeamBlockWidthForward >= 256) {
      if (tid < 128) { sA[tid] += sA[tid + 128]; } __syncthreads(); }
    if (InterTeamBlockWidthForward >= 128) {
      if (tid < 64) { sA[tid] += sA[tid + 64]; } __syncthreads(); }
    if (tid < 32) blockReduceAtBackwardInterTeamRR(sA, tid);
    
    if (tid == 0)  atomicAdd(&JVP[thetaIndex], sA[tid]);*/
  }

  UX[i][k] = UXi;
}


template <typename scalar_t> 
  __global__ void PlayIntraTeamTournamentsForThetaGrad(
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> UX,
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> G,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> C,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> S,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> JVP,
    const int Ntilde,
    const int dummyIndex,
    const int dMax)
{
  __shared__ scalar_t sAGridForBlock[IntraTeamBlockDepthForward][IntraTeamBlockWidthForward];
  const int tidY = threadIdx.y;
  scalar_t* sA = sAGridForBlock[tidY];
  
  // k is the column index of M and the row index of Uf, to set col of A
  const int tid = threadIdx.x;
  const int k = tid + blockDim.x*blockIdx.x; 
  if (k >= UX.size(1))
  {
    sA[tid] = 0;
    return;
  }

  int playerCountInBlock = IntraTeamBlockDepthForward *2;
  //if( k == 0 )  printf("init player count in block BACKWARD: %d\n",  playerCountInBlock);

  int blockStart = playerCountInBlock * blockIdx.y;

  //if( k == 0 )  printf("block start %d NTilde %d BACKWARD: \n",  blockStart, Ntilde);

  if (playerCountInBlock > Ntilde-blockStart)
  {
    playerCountInBlock = Ntilde-blockStart;
  }

  if (tidY*2 >= playerCountInBlock)
  {
    return;
  }

  const int N = UX.size(0);
  int i, j, thetaIndex;
  scalar_t cij, sij, UXi, UXj, newUXi, newUXj, Gi, Gj, newGi, newGj;

  //if( k == 0 )  printf("\ndummy id %d dMax %d  player count int block: %d\n", dummyIndex, dMax, playerCountInBlock);

  for (int tournamentStep=playerCountInBlock-2; tournamentStep>=0; tournamentStep--)
  {
    auto rowIndices = determineRowIndexPair(tidY, playerCountInBlock, tournamentStep);
    i = blockStart + rowIndices.first;
    j = blockStart + rowIndices.second;

    //if( k == 0 ) printf("blockId %d thread id %d tournamentStep: %d  i,j => %d %d INTRA NACKWARD got one! ->  S %.6f C %.6f \n ",blockIdx.y, tidY, tournamentStep, i, j, sij, cij);

    if (areRowIndicesOutOfRange(i, j, dummyIndex, dMax))
    {
      sA[tid] = 0;
      __syncthreads();
      continue;
    }

    thetaIndex = i*N - (i+2)*(i+1)/2 + j;
    cij = C[thetaIndex];
    sij = -1 * S[thetaIndex]; // Transpose of a Givens rotation has the signs of sij flipped


    Gi = G[i][k];
    Gj = G[j][k];

    newGi = Gi*cij - Gj*sij;
    G[i][k] = newGi;

    newGj = Gi*sij + Gj*cij;
    G[j][k] = newGj;

    UXi = UX[i][k];
    UXj = UX[j][k];

    newUXi = UXi*cij - UXj*sij;
    UX[i][k] = newUXi;

    newUXj = UXi*sij + UXj*cij;
    UX[j][k] = newUXj;

    sA[tid] = newUXi * newGj - newUXj * newGi;
    __syncthreads();

    // Reduce
    if (IntraTeamBlockWidthForward == 1024) {
      if (tid < 512) { sA[tid] += sA[tid + 512]; } __syncthreads(); }
    if (IntraTeamBlockWidthForward >= 512) {
      if (tid < 256) { sA[tid] += sA[tid + 256]; } __syncthreads(); }
    if (IntraTeamBlockWidthForward >= 256) {
      if (tid < 128) { sA[tid] += sA[tid + 128]; } __syncthreads(); }
    if (IntraTeamBlockWidthForward >= 128) {
      if (tid < 64) { sA[tid] += sA[tid + 64]; } __syncthreads(); }
    if (tid < 32) blockReduceAtBackwardIntraTeamRR(sA, tid);
    
    if (tid == 0)  atomicAdd(&JVP[thetaIndex], sA[tid]);
  }
}


void ScheduleInterTeamTournamentForThetaGrads(
  torch::Tensor C, 
  torch::Tensor S, 
  torch::Tensor UX,
  torch::Tensor G,
  torch::Tensor JVP,
  std::tuple<int, int, int> rotMatConstants)
{
  auto dMax = std::get<0>(rotMatConstants);
  auto dummyIndex = std::get<1>(rotMatConstants);
  auto Ntilde = std::get<2>(rotMatConstants);
  
  const int rotationCountPerRound = Ntilde/2;
  int teamCount = (Ntilde / InterTeamBlockDepthForward) + (Ntilde % InterTeamBlockDepthForward != 0);
  if (teamCount == 1)
  {
    return;
  }
  const int dummyTeamIndex = teamCount;
  incrementIfNotEven(teamCount);

  const dim3 blocks = prepareBlocksForTournament(UX.size(1), Ntilde, InterTeamBlockWidthForward, InterTeamBlockDepthForward);
  const dim3 threads(InterTeamBlockWidthForward, InterTeamBlockDepthForward);
  
  for (int tournamentStep=0; tournamentStep<=teamCount-2; tournamentStep++)
  {
    AT_DISPATCH_FLOATING_TYPES(
      C.type(),
      "rotMatForwardCuda",
      ([&]{ PlayTeamTournamentMatchForThetaGrad<scalar_t><<<blocks,threads>>>(
        UX.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
        G.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
        C.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
        S.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
        JVP.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
        dummyIndex, dMax, teamCount, dummyTeamIndex, tournamentStep);}));
  }
}

void ScheduleIntraTeamTournamentsForThetaGrads(
  torch::Tensor C, 
  torch::Tensor S, 
  torch::Tensor UX,
  torch::Tensor G,
  torch::Tensor JVP,
  std::tuple<int, int, int> rotMatConstants)
{
  auto dMax = std::get<0>(rotMatConstants);
  auto dummyIndex = std::get<1>(rotMatConstants);
  auto Ntilde = std::get<2>(rotMatConstants);

  const dim3 blocks = prepareBlocksForTournament(UX.size(1), Ntilde, IntraTeamBlockWidthForward, IntraTeamBlockDepthForward);
  const dim3 threads(IntraTeamBlockWidthForward, IntraTeamBlockDepthForward);

  AT_DISPATCH_FLOATING_TYPES(
    C.type(),
    "rotMatForwardCuda",
    ([&]{ PlayIntraTeamTournamentsForThetaGrad<scalar_t><<<blocks,threads>>>(
      UX.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
      G.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
      C.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
      S.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
      JVP.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
      Ntilde, dummyIndex, dMax);}));
}

std::pair<torch::Tensor, torch::Tensor> rotMatBackwardCudaTeamRR(
  torch::Tensor thetas,
  torch::Tensor UX,
  torch::Tensor G)
{
  auto constants = determineRotMatConstants(thetas.size(0), UX.size(0));
  auto C = torch::cos(thetas.detach());
  auto S = torch::sin(thetas.detach());

  auto thetasTensorOptions = torch::TensorOptions().dtype(thetas.dtype()).device(thetas.device());
  auto JVP = torch::zeros_like(thetas, thetasTensorOptions);

  ScheduleInterTeamTournamentForThetaGrads(C, S, UX, G, JVP, constants);
  //ScheduleIntraTeamTournamentsForThetaGrads(C, S, UX, G, JVP, constants);
  
  return std::make_pair(G, JVP);
}