#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <utility>
#include <functional>
#include <vector>

using namespace torch::indexing;

#define MaximumThreadsPerBlock 1024
#define WarpSize 32

#define ThreadsPerRowForward 128
#define ThreadsPerRowBackward 256

#define InterTeamRRThreadsPerBlockForward MaximumThreadsPerBlock
#define InterTeamBlockDepthForward (int)(InterTeamRRThreadsPerBlockForward/WarpSize)

// Current implementation dictates InterTeamBlockDepthForward to be 2 *IntraTeamBlockDepthForward
#define IntraTeamRRThreadsPerBlockForward (int)(MaximumThreadsPerBlock)
#define IntraTeamBlockDepthForward (int)(InterTeamBlockDepthForward/2)

// https://stackoverflow.com/questions/12626096/why-has-atomicadd-not-been-implemented-for-doubles
// https://stackoverflow.com/questions/37566987/cuda-atomicadd-for-doubles-definition-error
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(
  double* address, 
  double val)
{
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif

/************************* HELPER FUNCTIONS *********************************/

__device__ __forceinline__ std::pair<const int, const int> determineRowIndexPair(
  const int blockIndex,
  const int Ntilde,
  const int tournamentStep)
{
  const int n = Ntilde - 1;

  // Determine i,j from block index
  int i = blockIndex==0 ? 0 : blockIndex  + tournamentStep;
  if (i >= Ntilde)
  {
    i -= n;
  }

  int j = n - blockIndex + tournamentStep;
  if (j >= Ntilde)
  {
    j -= n;
  }

  return i > j ? std::make_pair(j,i) : std::make_pair(i,j);
}

__device__ __forceinline__ bool areRowIndicesOutOfRange(
  const int i, 
  const int j, 
  const int deadIndex, 
  const int dMax)
{
  // check if the coordinates are out of range or equal dummy coordinate (dummy exists when N is odd)
  return j == deadIndex || (i > dMax && j > dMax);
}


std::tuple<int, int, int> determineRotMatConstants(const int nThetas, const int N)
{
  auto dMax = N-1; // If nThetas == maxPairs
  if (nThetas < N*(N-1)/2)
  {
    dMax -= 1 + int(sqrt(1 - 4*(2*nThetas - N*(N-1)))) / 2;
  }

  // Handle odd N; in that case Ntilde is the even augmented dimension
  int deadIndex = -1;
  auto Ntilde = N;
  if (N % 2 != 0)
  {
    Ntilde += 1;
    deadIndex = Ntilde-1;
  }

  return std::make_tuple(dMax, deadIndex, Ntilde);
}


template <typename scalar_t>
  __device__ void warpReduceAtBackward(
    volatile scalar_t* sdata, 
    int tid)
{
  if (ThreadsPerRowBackward >= 64)  sdata[tid] += sdata[tid + 32];
  if (ThreadsPerRowBackward >= 32) sdata[tid] += sdata[tid + 16];
  if (ThreadsPerRowBackward >= 16) sdata[tid] += sdata[tid + 8];
  if (ThreadsPerRowBackward >= 8) sdata[tid] += sdata[tid + 4];
  if (ThreadsPerRowBackward >= 4) sdata[tid] += sdata[tid + 2];
  if (ThreadsPerRowBackward >= 2) sdata[tid] += sdata[tid + 1];
}

/************************* FORWARD PROPAGATION*******************************/
/****** USING THE CIRCLE METHOD FOR GENERATING THE ROUND ROBIN SEQUENCE *****/

 template <typename scalar_t>  
  __global__ void ApplyRoundRobinGivensRotationMatrix(
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> C,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> S,
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> U,
    const int deadIndex,
    const int Ntilde,
    const int dMax,
    const int tournamentStep)
{
  const int k = threadIdx.y + blockDim.y*blockIdx.y;
  if (k >= U.size(1))
  {
    return;
  }

  auto rowIndices = determineRowIndexPair(blockIdx.x, Ntilde, tournamentStep);
  const int i = rowIndices.first;
  const int j = rowIndices.second;
  if (areRowIndicesOutOfRange(i, j, deadIndex, dMax))
  {
    return;
  }

  const int N = U.size(0);
  const int thetaIndex = i*N - (i+2)*(i+1)/2 + j;
  const scalar_t cij = C[thetaIndex];
  const scalar_t sij = S[thetaIndex];

  // Apply Givens
  const scalar_t Ui = U[i][k];
  const scalar_t Uj = U[j][k];

  U[i][k] = Ui*cij - Uj*sij;
  U[j][k] = Ui*sij + Uj*cij;
}

torch::Tensor rotMatForwardCuda(torch::Tensor X, torch::Tensor thetas)
{
  const int N = X.size(0);
  auto rotMatConstants = determineRotMatConstants(thetas.size(0), N);
  auto dMax = std::get<0>(rotMatConstants);
  auto deadIndex = std::get<1>(rotMatConstants);
  auto Ntilde = std::get<2>(rotMatConstants);

  auto C = torch::cos(thetas.detach());
  auto S = torch::sin(thetas.detach());

  const int nBlocksY = X.size(1)/ThreadsPerRowForward + (X.size(1)% ThreadsPerRowForward != 0);
  const dim3 blocks(Ntilde / 2, nBlocksY);
  const dim3 threads(1, ThreadsPerRowForward);

  // The circle-method is used to generate round-robin sequences per block (equivalent to scheduling round-robin sports tournaments)
  // 'tournamentStep' refers to to the current turn of the tournament, where all updates are executed in parallel. There are n-1 steps
  for (int tournamentStep=Ntilde-2; tournamentStep>=0; tournamentStep--)
  {
    AT_DISPATCH_FLOATING_TYPES(
      thetas.type(),
      "rotMatForwardCuda",
      ([&]{
        ApplyRoundRobinGivensRotationMatrix<scalar_t><<<blocks,threads>>>(
          C.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
          S.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
          X.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
          deadIndex,Ntilde, dMax, tournamentStep);
      }));
  }

  return X;
}

/************************* BACKWARD PROPAGATION******************************/
// USING THE CIRCLE ROUND ROBIN TOURNAMENT FOR SEQUENCING GIVENS ROTATIONS

template <typename scalar_t> 
  __global__ void CalculateRoundRobinGivensThetaJVPs(
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> UX,
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> G,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> C,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> S,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> JVP,
    const int deadIndex,
    const int Ntilde,
    const int dMax,
    const int tournamentStep)
{
  __shared__ scalar_t sA[ThreadsPerRowBackward];

  // k is the column index of M and the row index of Uf, to set col of A
  const int tid = threadIdx.y;
  const int k = tid + blockDim.y*blockIdx.y;
  if (k >= UX.size(1))
  {
    sA[tid] = 0;
    return;
  }

  auto rowIndices = determineRowIndexPair(blockIdx.x, Ntilde, tournamentStep);
  int i = rowIndices.first;
  int j = rowIndices.second;
  if (areRowIndicesOutOfRange(i, j, deadIndex, dMax))
  {
    sA[tid] = 0;
    return;
  }

   __syncthreads();
  const int N = UX.size(0);
  const int thetaIndex = i*N - (i+2)*(i+1)/2 + j;
  const scalar_t cij = C[thetaIndex];
  const scalar_t sij = -1 * S[thetaIndex]; // Transpose of a Givens rotation has the signs of sij flipped

  // Apply Givens Transpose to G
  const scalar_t Gi = G[i][k];
  const scalar_t Gj = G[j][k];

  const scalar_t newGi = Gi*cij - Gj*sij;
  G[i][k] = newGi;

  const scalar_t newGj = Gi*sij + Gj*cij;
  G[j][k] = newGj;

  // Repeat for UX
  const scalar_t UXi = UX[i][k];
  const scalar_t UXj = UX[j][k];

  const scalar_t newUXi = UXi*cij - UXj*sij;
  UX[i][k] = newUXi;

  const scalar_t newUXj = UXi*sij + UXj*cij;
  UX[j][k] = newUXj;

  sA[tid] = newUXi * newGj - newUXj * newGi;
  __syncthreads();

  // Reduce
  if (ThreadsPerRowBackward == 1024) {
    if (tid < 512) { sA[tid] += sA[tid + 512]; } __syncthreads();}
  if (ThreadsPerRowBackward >= 512) {
    if (tid < 256) { sA[tid] += sA[tid + 256]; } __syncthreads();}
  if (ThreadsPerRowBackward >= 256) {
    if (tid < 128) { sA[tid] += sA[tid + 128]; } __syncthreads(); }
  if (ThreadsPerRowBackward >= 128) {
    if (tid < 64) { sA[tid] += sA[tid + 64]; } __syncthreads(); }
  if(tid < 32) warpReduceAtBackward(sA, tid);
  
  if (tid == 0)  atomicAdd(&JVP[thetaIndex], sA[tid]);
}

std::pair<torch::Tensor, torch::Tensor> rotMatBackwardCuda(
  torch::Tensor thetas,
  torch::Tensor UX,
  torch::Tensor G)
{
  auto N = UX.size(0);
  auto rotMatConstants = determineRotMatConstants(thetas.size(0), N);
  auto dMax = std::get<0>(rotMatConstants);
  auto deadIndex = std::get<1>(rotMatConstants);
  auto Ntilde = std::get<2>(rotMatConstants);

  auto C = torch::cos(thetas.detach());
  auto S = torch::sin(thetas.detach());

  auto thetasTensorOptions = torch::TensorOptions().dtype(thetas.dtype()).device(thetas.device());
  auto JVP = torch::zeros_like(thetas, thetasTensorOptions);

  const int nBlocksY = N/ThreadsPerRowBackward + (N % ThreadsPerRowBackward != 0);
  const dim3 blocks(Ntilde / 2, nBlocksY);
  const dim3 threads(1, ThreadsPerRowBackward);

  // The circle-method is used to generate round-robin sequences per block (equivalent to scheduling round-robin sports tournaments)
  // 'tournamentStep' refers to to the current turn of the tournament, where all updates are executed in parallel. here are n-1 steps
  for (int tournamentStep=0; tournamentStep<=Ntilde-2; tournamentStep++)
  {
    AT_DISPATCH_FLOATING_TYPES(
      thetas.type(),
      "rotMatBackwardCuda",
      ([&]{
        CalculateRoundRobinGivensThetaJVPs<scalar_t><<<blocks,threads>>>(
          UX.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
          G.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
          C.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
          S.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
          JVP.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
          deadIndex, Ntilde, dMax, tournamentStep);
      }));
  }
  
  return std::make_pair(G, JVP);
}

/************************* FORWARD PROPAGATION*******************************/
// USING THE TEAM ROUND ROBIN TOURNAMENT FOR SEQUENCING GIVENS ROTATIONS

template <typename scalar_t>
  __global__ void PlayIndividualTournamentWithinTeams(
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> C,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> S,
    at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> X,
    const int Ntilde,
    const int deadIndex,
    const int dMax)
{
  // If transpose k works on rows; otherwise on columns
  const int tid = threadIdx.x;
  
  const int k = tid + blockDim.x*blockIdx.x; 
  if (k >= X.size(1) || k*2 >= Ntilde)
  {
    return;
  }

  int playerCountInBlock = IntraTeamBlockDepthForward *2;
  if (playerCountInBlock > Ntilde)
  {
    playerCountInBlock = Ntilde;
  }

  int blockStart = playerCountInBlock * blockIdx.y;
  const int N = X.size(0);

  int i, j, thetaIndex;
  scalar_t cij, sij, Xi, Xj;
  for (int tournamentStep=0; tournamentStep<=playerCountInBlock-2; tournamentStep++)
  {
    auto rowIndices = determineRowIndexPair(threadIdx.y, playerCountInBlock, tournamentStep);
    i = blockStart + rowIndices.first;
    j = blockStart + rowIndices.second;
    
    if (areRowIndicesOutOfRange(i, j, deadIndex, dMax))
    {
      __syncthreads();
      continue;
    }

    thetaIndex = i*N - (i+2)*(i+1)/2 + j;
    cij = C[thetaIndex];
    sij = S[thetaIndex];
    
    Xi = X[i][k];
    Xj = X[j][k];

    X[i][k] = Xi*cij - Xj*sij;
    X[j][k] = Xi*sij + Xj*cij;
    __syncthreads();
  }
}

/*
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> C,
    at::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> S,
*/
template <typename scalar_t> __global__ void PlayTeamTournamentMatch(
  const scalar_t* const __restrict__ C,
  const scalar_t* const __restrict__ S,
  at::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> U,
  const int deadIndex,
  const int dMax,
  const int teamCount,
  const int tournamentStep)
{
  // If transpose k works on rows; otherwise on columns
  const int col = threadIdx.x + blockDim.x*blockIdx.x;
  if (col >= U.size(1))
  {
    return;
  }

  // rn 2*blockDepth-1 and ntilde-1 are eqaul, need to change what we pass to determineRowIndexPAir instead of ntilde
  auto rowIndices = determineRowIndexPair(blockIdx.y, teamCount, tournamentStep);
  
  const int playerCountPerTeam = blockDim.y;
  const int i =  playerCountPerTeam * rowIndices.first + threadIdx.y; // home team
  
  const int jStart = playerCountPerTeam * rowIndices.second;
  const int jEnd = jStart + playerCountPerTeam;
  int j = jStart + threadIdx.y;
  
  const int N = U.size(0);
  scalar_t Ui = U[i][col];
  for (int step =0; step < playerCountPerTeam; step++, j++)
  {
    if (j >= jEnd)
    {
      j -= playerCountPerTeam;
    }
    
    if (areRowIndicesOutOfRange(i, j, deadIndex, dMax))
    {
      __syncthreads();
      continue;
    }

    const int thetaIndex = i*N - (i+2)*(i+1)/2 + j;
    const scalar_t cij = C[thetaIndex];
    const scalar_t sij = S[thetaIndex];

    // Apply Givens: Update U's offsets
    const scalar_t Uj = U[j][col];

    // must update uj before updating ui
    U[j][col] = Ui*sij + Uj*cij;
    Ui = Ui*cij - Uj*sij;

    __syncthreads();
  }

  U[i][col] = Ui;
}

torch::Tensor rotMatForwardCudaTeamRR(torch::Tensor X, torch::Tensor thetas)
{
  const int N = X.size(0);
  auto rotMatConstants = determineRotMatConstants(thetas.size(0), N);
  auto dMax = std::get<0>(rotMatConstants);
  auto deadIndex = std::get<1>(rotMatConstants);
  auto Ntilde = std::get<2>(rotMatConstants);

  auto C = torch::cos(thetas.detach());
  auto S = torch::sin(thetas.detach());

  int threadsWithIdenticalWork = IntraTeamRRThreadsPerBlockForward/IntraTeamBlockDepthForward;
  const dim3 threads(threadsWithIdenticalWork, IntraTeamBlockDepthForward);
  
  int nBlocksX = (N/threadsWithIdenticalWork) + (N %threadsWithIdenticalWork != 0); // 1
  int nBlocksY = ((Ntilde / 2)/IntraTeamBlockDepthForward) + ((Ntilde / 2) % IntraTeamBlockDepthForward != 0); // 2
  const dim3 blocks(nBlocksX, nBlocksY);

  // The circle-method is used to generate round-robin sequences per block (equivalent to scheduling round-robin sports tournaments)
  // 'tournamentStep' refers to to the current turn of the tournament, where all updates are executed in parallel. There are n-1 steps
  AT_DISPATCH_FLOATING_TYPES(
    thetas.type(),
    "rotMatForwardCuda",
    ([&]{ PlayIndividualTournamentWithinTeams<scalar_t><<<blocks,threads>>>(
      C.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
      S.packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
      X.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
      Ntilde, deadIndex, dMax);}));
  
  // Does the entire tournament fit into a single block?
  if (nBlocksY == 1)
  {
    return X;
  }

  threadsWithIdenticalWork = InterTeamRRThreadsPerBlockForward/InterTeamBlockDepthForward;
  const dim3 threads2(threadsWithIdenticalWork, InterTeamBlockDepthForward);
  
  const int nBlocksX2 = (N/threadsWithIdenticalWork) + (N%threadsWithIdenticalWork != 0);
  const int nBlocksY2 = ((Ntilde / 2)/InterTeamBlockDepthForward) + ((Ntilde / 2)% InterTeamBlockDepthForward != 0);
  const dim3 blocks2(nBlocksX2, nBlocksY2);

  const int teamCount = (Ntilde/InterTeamBlockDepthForward) + (Ntilde%InterTeamBlockDepthForward != 0);
  for (int tournamentStep=teamCount-2; tournamentStep>=0; tournamentStep--)
  {
    AT_DISPATCH_FLOATING_TYPES(
      thetas.type(),
      "rotMatForwardCuda",
      ([&]{ PlayTeamTournamentMatch<scalar_t><<<blocks2,threads2>>>(
        C.data_ptr<scalar_t>(),
        S.data_ptr<scalar_t>(),
        X.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
        deadIndex, dMax, teamCount, tournamentStep);}));
  }

  return X;
}

/************************* BACKWARD PROPAGATION*******************************/
// USING THE TEAM ROUND ROBIN TOURNAMENT FOR SEQUENCING GIVENS ROTATIONS